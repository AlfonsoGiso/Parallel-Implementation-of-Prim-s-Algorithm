/*
 * Course: High Performance Computing 2023/2024
 *
 * Lecturer: Francesco Moscato  fmoscato@unisa.it * 
 * Student: Giso  Alfonso  0622701842   a.giso@studenti.unisa.it
 * 
 * Provide a parallell version of the Prim's algorithm to find the minimum MST of a graph.
 * 
 * The implementation MUST use an hibrid  of CUDA and openMP.
 * 
 * This code was created from the following, visible through the link:
 * 
 * https://www.geeksforgeeks.org/prims-minimum-spanning-tree-mst-greedy-algo-5/
 * 
 * Copyright (C) 2024  Alfonso Giso
 * 
 * This file is part of CommonAssignmentMPIOpenMP
 * 
 * CommonAssignmentMPIOpenMP is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * CommonAssignmentMPIOpenMP is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with CommonAssignmentMPIOpenMP . If not, see <http://www.gnu.org/licenses/>.
 * 
*/

#include <hip/hip_runtime.h>
#include <limits.h>
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define BLOCK_SIZE 256

typedef struct Edge {
    int start, end, weight;
} Edge;


__device__ int findEdgeWithMinKey(bool *mstSet, Edge *edges, int edgeCount, int rows) {
    int min = INT_MAX, min_index = -1;
    #pragma omp parallel for
    for (int i = 0; i < edgeCount; i++) {
        // Verifica se almeno uno dei nodi collegati è già nell'insieme MST
        if ((mstSet[edges[i].start] || mstSet[edges[i].end]) &&
            edges[i].weight < min && 
            ((mstSet[edges[i].start] && !mstSet[edges[i].end]) || 
             (mstSet[edges[i].end] && !mstSet[edges[i].start]))) 
             {
            min = edges[i].weight;
            min_index = i;
        }
    }
    //printf("Sono il thread: %d e l'indice minimo che ho trovato è:%d\n",threadIdx.x,min_index);

    return min_index;
}


__device__ int findGlobalMinIndex(Edge *edges, int *localMinIndices, int gridDim, int blockDim) {
    int globalMinIndex = -1;
    int globalMinWeight = INT_MAX;
    for (int i = 0; i <blockDim; ++i) {
        int localIndex = localMinIndices[i % blockDim];
        if (localIndex != -1) {
            int localWeight = edges[localIndex].weight;
            if (localWeight < globalMinWeight) {
                globalMinWeight = localWeight;
                globalMinIndex = localIndex;
            }
        }
    }

    return globalMinIndex;
}




__global__ void primMSTKernel(bool *mstSet, Edge *edges, int rows, int edgeCount, 
int *minIndices,int *d_globalMinWeight,int *d_currentIndex,int i) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Calcolo di quanti edge deve gestire ogni thread
    int edgesPerThread = (edgeCount + blockDim.x - 1) / blockDim.x;

    // Calcolo degli indici di inizio e fine per gli edge assegnati al thread
    int startIdx = tid * edgesPerThread;
    int endIdx = min(startIdx + edgesPerThread, edgeCount);

    // Variabile condivisa per l'indice del minimo locale all'interno del blocco
    __shared__ int blockMinIndex[BLOCK_SIZE];

    // Inizializza blockMinIndex con un valore non valido
    blockMinIndex[threadIdx.x] = -1;
    __syncthreads();

    // Ciascun thread trova il minimo locale solo nei suoi edge
    int minIndex = -1;
    int minWeight = INT_MAX;
    for (int i = startIdx; i < endIdx; ++i) {
        // Verifica se almeno uno dei nodi collegati è già nell'insieme MST
        if ((mstSet[edges[i].start] || mstSet[edges[i].end]) &&
            edges[i].weight < minWeight &&
            ((mstSet[edges[i].start] && !mstSet[edges[i].end]) ||
             (mstSet[edges[i].end] && !mstSet[edges[i].start]))) {
            minWeight = edges[i].weight;
            minIndex = i;
        }
    }
  // Ogni thread salva il suo indice di minimo locale nell'array minIndices
    minIndices[threadIdx.x] = minIndex;
    __syncthreads();

   //Il thread 0 di ciascun blocco trova il minimo tra gli indici di minimo locale
    if (threadIdx.x == 0) {
        int blockMinIndex = findGlobalMinIndex(edges, minIndices, gridDim.x, blockDim.x);

        // Se è stato trovato un indice valido, aggiorna il minimo globale e l'indice corrente
        if (blockMinIndex != -1) {
            atomicMin(d_globalMinWeight, edges[blockMinIndex].weight);
            *d_currentIndex = blockMinIndex;
        }
    }
    __syncthreads();

    // Il thread 0 di ciascun blocco contribuisce al calcolo del minimo globale
    if (threadIdx.x == 0 && blockMinIndex[0] != -1) {
        atomicMin(d_globalMinWeight, edges[blockMinIndex[0]].weight);
    }
    __syncthreads();

    // Il thread 0 del blocco con il minimo locale aggiorna l'indice corrente
    if (minWeight == edges[blockMinIndex[0]].weight && threadIdx.x == 0) {
        printf("Min Index:%d\n",blockMinIndex[0]);
        *d_currentIndex = blockMinIndex[0];
    }
    __syncthreads();
}







int main(int argc, char **argv) {
    char *inputFilePath = argv[1];

    FILE *file = fopen(inputFilePath, "r");
    if (file == NULL) {
        perror("Errore nell'apertura del file");

    }
    int rows= atoi(argv[2]);

    int total_elements = rows * rows;
    int *matrix = (int *)malloc(total_elements * sizeof(int));

    for (int i = 0; i < total_elements; i++) {
         int result = fscanf(file, "%d", &matrix[i]);
    if (result != 1) {
        perror("Errore nella lettura della matrice");
        return 1;
    }
    }

    fclose(file);

    struct Edge *edges = (struct Edge *)malloc((rows * (rows - 1) / 2) * sizeof(struct Edge));
    
    int edgeCount = 0;

    for (int i = 0; i < rows; i++) {
        for (int j = i + 1; j < rows; j++) {
            if (matrix[i * rows + j] != 0) {
                edges[edgeCount].start = i;
                edges[edgeCount].end = j;
                edges[edgeCount].weight = matrix[i * rows + j];
                edgeCount++;
            }
        }
    }
    printf("Edge count:%d\n",edgeCount);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    float elapsed;
    Edge *d_edges;
    bool *d_mstSet;
    int *d_minWeights;
    bool mstSet[rows];
    int h_minWeight=0;
    int h_globalMinWeight;
    int *d_globalMinWeight;
    int h_currentIndex;
    int *d_currentIndex;
    hipMalloc((void **)&d_currentIndex,sizeof(int));
    hipMalloc((void **)&d_globalMinWeight, sizeof(int));
    hipMalloc((void **)&d_edges, edgeCount * sizeof(Edge));
    hipMalloc((void **)&d_minWeights, BLOCK_SIZE * sizeof(int));
    hipMalloc((void **)&d_mstSet, rows * sizeof(bool));

    

    // Inizializza la somma del peso dell'MST a 0
    int totalWeight = 0;
    
    for (int j = 0; j < rows; j++) {
            mstSet[j] = false;
        }
        mstSet[0] = true;
    int blockCount = (rows -1 + BLOCK_SIZE -1 ) / BLOCK_SIZE;
    printf("Block count:%d\n",blockCount);
    float times[10];
    
    for(int j =0;j<10;j++){
    hipEventRecord(start,0);
    #pragma omp parallel sections
{

    #pragma omp section
    {
        hipMemcpy(d_edges, edges, edgeCount * sizeof(Edge), hipMemcpyHostToDevice);
    }

    #pragma omp section
    {
        hipMemcpy(d_mstSet, mstSet, rows * sizeof(bool), hipMemcpyHostToDevice);
    }

    #pragma omp section
    {
        hipMemcpy(d_minWeights, &h_minWeight, sizeof(int), hipMemcpyHostToDevice);
    }

}

    
    // Esegui il kernel più volte finché non vengono inseriti tutti i vertici in mstSet
    for (int i = 0; i < rows-1; i++) {

        primMSTKernel<<<blockCount, BLOCK_SIZE>>>(d_mstSet, d_edges, rows, edgeCount, 
        d_minWeights,d_globalMinWeight,d_currentIndex,i);
        
        //Aggiorno il peso totale dell'MST
        hipMemcpy(&h_globalMinWeight, d_globalMinWeight, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&h_currentIndex,d_currentIndex,sizeof(int),hipMemcpyDeviceToHost);
        mstSet[edges[h_currentIndex].start]=true;
        mstSet[edges[h_currentIndex].end]=true;
        totalWeight += edges[h_currentIndex].weight;
        hipMemcpy(d_mstSet, mstSet, rows * sizeof(bool), hipMemcpyHostToDevice);
        hipDeviceSynchronize();

    }
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    // execution time between events in ms
    hipEventElapsedTime(&elapsed, start, stop);
    elapsed = elapsed/1000.f; // convert to seconds
    times[j]=elapsed;
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
    float total;

    for(int i=0;i<10;i++){
        total+=times[i];
    }
    printf("CUDA Time : %f", (double)(total/10));

    hipFree(d_edges);
    hipFree(d_mstSet);
    hipFree(d_minWeights);
    hipFree(d_globalMinWeight);

    free(edges);
    free(matrix);


    return 0;
}
